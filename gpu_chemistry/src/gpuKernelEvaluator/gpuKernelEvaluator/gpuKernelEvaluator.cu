#include "hip/hip_runtime.h"
#include "gpuKernelEvaluator.H"

#include <iostream>

#include "cuda_host_dev.H"
#include "gpuBuffer.H"
#include "host_device_vectors.H"

#include <thrust/execution_policy.h>
#include <thrust/extrema.h> //min_element
#include <thrust/host_vector.h>

namespace FoamGpu {

GpuKernelEvaluator::GpuKernelEvaluator(
    gLabel                          nEqns,
    gLabel                          nSpecie,
    const std::vector<gpuThermo>&   thermos,
    const std::vector<gpuReaction>& reactions,
    gpuODESolverInputs              odeInputs)
    : nEqns_(nEqns)
    , nSpecie_(nSpecie)
    , nReactions_(gLabel(reactions.size()))
    , thermosReactions_(thermos, reactions)
    , system_(nEqns_,
              gLabel(reactions.size()),
              thermosReactions_.thermos(),
              thermosReactions_.reactions())
    , solver_(make_gpuODESolver(system_, odeInputs)) {}

template <class S1, class S2, class S3, class S4> struct singleCell {

    singleCell(gScalar      deltaT,
               gLabel       nSpecie,
               S1           deltaTChem,
               S2           Yvf,
               S3           Jss,
               S4           buffer,
               gpuODESolver ode)
        : deltaT_(deltaT)
        , nSpecie_(nSpecie)
        , deltaTChem_(deltaTChem)
        , Yvf_(Yvf)
        , Jss_(Jss)
        , buffer_(buffer)
        , ode_(ode) {}

    CUDA_HOSTDEV void operator()(gLabel celli) const {
        auto Y = mdspan<gScalar, 1>(&Yvf_(celli, 0), extents<1>{nSpecie_ + 2});
        auto J = mdspan<gScalar, 2>(&Jss_(celli, 0, 0),
                                    extents<2>{nSpecie_ + 2, nSpecie_ + 2});

        // Initialise time progress
        gScalar timeLeft = deltaT_;

        constexpr gLabel li = 0;

        // Calculate the chemical source terms
        while (timeLeft > gpuSmall) {
            gScalar dt = timeLeft;

            ode_.solve(0, dt, Y, li, deltaTChem_[celli], J, buffer_[celli]);

            for (int i = 0; i < nSpecie_; i++) { Y[i] = std::max(0.0, Y[i]); }

            timeLeft -= dt;
        }
    }

    gScalar      deltaT_;
    gLabel       nSpecie_;
    S1           deltaTChem_;
    S2           Yvf_;
    S3           Jss_;
    S4           buffer_;
    gpuODESolver ode_;
};

std::pair<std::vector<gScalar>, std::vector<gScalar>>
GpuKernelEvaluator::computeYNew(gScalar                     deltaT,
                                gScalar                     deltaTChemMax,
                                const std::vector<gScalar>& deltaTChem,
                                const std::vector<gScalar>& Y) const {

    const gLabel nCells = deltaTChem.size();

    // Convert fields from host to device
    auto ddeltaTChem_arr = toDeviceVector(deltaTChem);
    auto dYvf_arr        = toDeviceVector(Y);
    auto ddeltaTChem     = make_mdspan(ddeltaTChem_arr, extents<1>{nCells});
    auto dYvf            = make_mdspan(dYvf_arr, extents<2>{nCells, nEqns_});

    device_vector<gScalar> Js(nCells * nEqns_ * nEqns_, 0.0);

    auto Jss = make_mdspan(Js, extents<3>{nCells, nEqns_, nEqns_});

    device_vector<gpuBuffer> buffer_arr =
        host_vector<gpuBuffer>(nCells, gpuBuffer(nSpecie_));

    auto buffer = make_mdspan(buffer_arr, extents<1>{nCells});

    singleCell op(deltaT, nSpecie_, ddeltaTChem, dYvf, Jss, buffer, solver_);

    thrust::for_each(thrust::device,
                     thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(nCells),
                     op);

    return std::make_pair(toStdVector(dYvf_arr), toStdVector(ddeltaTChem_arr));
}

std::tuple<std::vector<gScalar>, std::vector<gScalar>, gScalar>
GpuKernelEvaluator::computeRR(gScalar                    deltaT,
                              gScalar                    deltaTChemMax,
                              const std::vector<gScalar> rho,
                              const std::vector<gScalar> deltaTChem,
                              const std::vector<gScalar> Y) const {

    const gLabel nCells = rho.size();

    auto pair          = computeYNew(deltaT, deltaTChemMax, deltaTChem, Y);
    auto YNew_arr      = std::get<0>(pair);
    auto deltaTChemNew = std::get<1>(pair);

    auto YNew = make_mdspan(YNew_arr, extents<2>{nCells, nEqns_});
    auto Y0   = make_mdspan(Y, extents<2>{nCells, nEqns_});

    std::vector<gScalar> RR_arr(nCells * nSpecie_);
    auto                 RR = make_mdspan(RR_arr, extents<2>{nCells, nSpecie_});

    for (gLabel j = 0; j < nCells; ++j) {
        for (gLabel i = 0; i < nSpecie_; ++i) {

            RR(j, i) = rho[j] * (YNew(j, i) - Y0(j, i)) / deltaT;
        }
    }

    gScalar deltaTMin =
        *std::min_element(deltaTChemNew.begin(), deltaTChemNew.end());

    for (auto& e : deltaTChemNew) { e = std::min(e, deltaTChemMax); }

    return std::make_tuple(RR_arr, deltaTChemNew, deltaTMin);
}

} // namespace FoamGpu