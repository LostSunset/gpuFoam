#include "hip/hip_runtime.h"
#include "gpuKernelEvaluator.H"

#include <iostream>

#include "cuda_host_dev.H"
#include "host_device_vectors.H"
#include "gpuBuffer.H"

#include <thrust/execution_policy.h>
#include <thrust/extrema.h> //min_element
#include <thrust/host_vector.h>

namespace FoamGpu {


template<class S1, class S2, class S3, class S4>
struct singleCell{

    singleCell(gScalar      deltaT,
               gLabel       nSpecie,
               S1           deltaTChem,
               S2           Yvf,
               S3           Jss,
               S4           buffer,
               gpuODESolver ode)
        : deltaT_(deltaT)
        , nSpecie_(nSpecie)
        , deltaTChem_(deltaTChem)
        , Yvf_(Yvf)
        , Jss_(Jss)
        , buffer_(buffer)
        , ode_(ode){}

    CUDA_HOSTDEV void operator()(gLabel celli) const {
        auto Y = mdspan<gScalar, 1>(&Yvf_(celli, 0), extents<1>{nSpecie_ + 2});
        auto J = mdspan<gScalar, 2>(&Jss_(celli, 0, 0),
                                    extents<2>{nSpecie_ + 2, nSpecie_ + 2});

        // Initialise time progress
        gScalar timeLeft = deltaT_;

        constexpr gLabel li = 0;

        // Calculate the chemical source terms
        while (timeLeft > gpuSmall) {
            gScalar dt = timeLeft;

            ode_.solve(0, dt, Y, li, deltaTChem_[celli], J, buffer_[celli]);

            for (int i = 0; i < nSpecie_; i++) { Y[i] = std::max(0.0, Y[i]); }

            timeLeft -= dt;
        }
        }

    gScalar deltaT_;
    gLabel nSpecie_;
    S1 deltaTChem_;
    S2 Yvf_;
    S3 Jss_;
    S4 buffer_;
    gpuODESolver ode_;
};


template <class Op>
__global__ void cuda_kernel(Op op, gLabel nCells) {

    int celli = blockIdx.x * blockDim.x + threadIdx.x;
    if (celli < nCells)
    {
        op(celli);
    }
}


std::pair<std::vector<gScalar>, std::vector<gScalar>>
GpuKernelEvaluator::computeYNew(gScalar                     deltaT,
                                gScalar                     deltaTChemMax,
                                const std::vector<gScalar>& deltaTChem,
                                const std::vector<gScalar>& Y) {

    const gLabel nCells = deltaTChem.size();

    // Convert thermos and reactions from host to device
    const auto dThermos   = toDeviceVector(hThermos_);
    const auto dReactions = toDeviceVector(hReactions_);

    gpuODESystem odeSystem(nEqns_,
                           dReactions.size(),
                           thrust::raw_pointer_cast(dThermos.data()),
                           thrust::raw_pointer_cast(dReactions.data()));

    gpuODESolver ode = make_gpuODESolver(odeSystem, odeInputs_);

    // Convert fields from host to device
    auto ddeltaTChem_arr = toDeviceVector(deltaTChem);
    auto dYvf_arr        = toDeviceVector(Y);
    auto ddeltaTChem     = make_mdspan(ddeltaTChem_arr, extents<1>{nCells});
    auto dYvf            = make_mdspan(dYvf_arr, extents<2>{nCells, nEqns_});

    device_vector<gScalar> Js(nCells * nEqns_ * nEqns_, 0.0);

    auto Jss = make_mdspan(Js, extents<3>{nCells, nEqns_, nEqns_});

    device_vector<gpuBuffer> buffer_arr =
        host_vector<gpuBuffer>(nCells, gpuBuffer(nSpecie_));

    auto buffer = make_mdspan(buffer_arr, extents<1>{nCells});

    singleCell op(deltaT, nSpecie_, ddeltaTChem, dYvf, Jss, buffer, ode);
    thrust::for_each
    (
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(nCells),
        op
    );


    /*
    singleCell op(deltaT, nSpecie_, ddeltaTChem, dYvf, Jss, buffer, ode);
    gLabel NTHREADS = 32;
    gLabel NBLOCKS  = (nCells + NTHREADS - 1) / NTHREADS;
    cuda_kernel<<<NBLOCKS, NTHREADS>>>(op, nCells);
    */

    return std::make_pair(toStdVector(dYvf_arr), toStdVector(ddeltaTChem_arr));
}

std::tuple<std::vector<gScalar>, std::vector<gScalar>, gScalar>
GpuKernelEvaluator::computeRR(gScalar                    deltaT,
                              gScalar                    deltaTChemMax,
                              const std::vector<gScalar> rho,
                              const std::vector<gScalar> deltaTChem,
                              const std::vector<gScalar> Y) {

    const gLabel nCells = rho.size();

    auto pair = computeYNew(deltaT, deltaTChemMax, deltaTChem, Y);
    auto YNew_arr = std::get<0>(pair);
    auto deltaTChemNew = std::get<1>(pair);

    auto YNew = make_mdspan(YNew_arr, extents<2>{nCells, nEqns_});
    auto Y0 = make_mdspan(Y, extents<2>{nCells, nEqns_});

    std::vector<gScalar> RR_arr(nCells * nSpecie_);
    auto                 RR = make_mdspan(RR_arr, extents<2>{nCells, nSpecie_});

    for (gLabel j = 0; j < nCells; ++j) {
        for (gLabel i = 0; i < nSpecie_; ++i) {

            RR(j, i) = rho[j] * (YNew(j, i) - Y0(j, i)) / deltaT;
        }
    }

    gScalar deltaTMin =
        *std::min_element(deltaTChemNew.begin(), deltaTChemNew.end());

    for (auto& e : deltaTChemNew) { e = std::min(e, deltaTChemMax); }

    return std::make_tuple(RR_arr, deltaTChemNew, deltaTMin);
}

} // namespace FoamGpu