#include "hip/hip_runtime.h"
#include "gpuKernelEvaluator.H"

#include <iostream>

#include "cuda_host_dev.H"
#include "gpuBuffer.H"
#include "gpuMemoryResource.H"
#include "host_device_vectors.H"

#include <thrust/execution_policy.h>
#include <thrust/extrema.h> //min_element
#include <thrust/host_vector.h>

#include "variant.hpp"
#include <thrust/device_malloc_allocator.h>

namespace FoamGpu {

using labelAllocator  = thrust::device_malloc_allocator<gLabel>;
using scalarAllocator = thrust::device_malloc_allocator<gScalar>;
using memoryResource_t =
    FoamGpu::gpuMemoryResource<labelAllocator, scalarAllocator>;

GpuKernelEvaluator::GpuKernelEvaluator(
    gLabel                          nEqns,
    gLabel                          nSpecie,
    const std::vector<gpuThermo>&   thermos,
    const std::vector<gpuReaction>& reactions,
    gpuODESolverInputs              odeInputs)
    : nEqns_(nEqns)
    , nSpecie_(nSpecie)
    , nReactions_(gLabel(reactions.size()))
    , thermosReactions_(thermos, reactions)
    , system_(nEqns_,
              gLabel(reactions.size()),
              thermosReactions_.thermos(),
              thermosReactions_.reactions())
    , solver_(make_gpuODESolver(system_, odeInputs))
    , inputs_(odeInputs) {}

template <class ODE> struct singleCell {

    singleCell(gScalar              deltaT,
               gLabel               nSpecie,
               mdspan<gScalar, 1>   deltaTChem,
               mdspan<gScalar, 2>   Yvf,
               mdspan<gpuBuffer, 1> buffer,
               ODE                  ode)
        : deltaT_(deltaT)
        , nSpecie_(nSpecie)
        , deltaTChem_(deltaTChem)
        , Yvf_(Yvf)
        , buffer_(buffer)
        , ode_(ode) {}

    CUDA_HOSTDEV void operator()(gLabel celli) const {
        auto Y = mdspan<gScalar, 1>(&Yvf_(celli, 0), extents<1>{nSpecie_ + 2});

        // Initialise time progress
        gScalar timeLeft = deltaT_;

        constexpr gLabel li = 0;

        // Calculate the chemical source terms
        while (timeLeft > gpuSmall) {
            gScalar dt = timeLeft;

            ode_.solve(0, dt, Y, li, deltaTChem_[celli], buffer_[celli]);

            for (int i = 0; i < nSpecie_; i++) { Y[i] = std::max(0.0, Y[i]); }

            timeLeft -= dt;
        }
    }

    gScalar              deltaT_;
    gLabel               nSpecie_;
    mdspan<gScalar, 1>   deltaTChem_;
    mdspan<gScalar, 2>   Yvf_;
    mdspan<gpuBuffer, 1> buffer_;
    ODE                  ode_;
};

std::pair<std::vector<gScalar>, std::vector<gScalar>>
GpuKernelEvaluator::computeYNew(gScalar                     deltaT,
                                gScalar                     deltaTChemMax,
                                const std::vector<gScalar>& deltaTChem,
                                const std::vector<gScalar>& Y) const {

    const gLabel nCells = deltaTChem.size();

    // Convert fields from host to device
    auto ddeltaTChem_arr = toDeviceVector(deltaTChem);
    auto dYvf_arr        = toDeviceVector(Y);
    auto ddeltaTChem     = make_mdspan(ddeltaTChem_arr, extents<1>{nCells});
    auto dYvf            = make_mdspan(dYvf_arr, extents<2>{nCells, nEqns_});

    //device_vector<gScalar> Js(nCells * nEqns_ * nEqns_, 0.0);

    //auto Jss = make_mdspan(Js, extents<3>{nCells, nEqns_, nEqns_});

    memoryResource_t mr(nCells, nSpecie_);
    auto             buffers     = toDeviceVector(splitToBuffers(mr));
    auto             buffer_span = make_mdspan(buffers, extents<1>{nCells});

    /*
    if (inputs_.name == "Rosenbrock23") {

        gpuRosenbrock23<gpuODESystem> ode(system_, inputs_);
        singleCell                    op(
            deltaT, nSpecie_, ddeltaTChem, dYvf, Jss, buffer_span, ode);
        thrust::for_each(thrust::device,
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(nCells),
                         op);
    } else {
        gpuRosenbrock34<gpuODESystem> ode(system_, inputs_);
        singleCell                    op(
            deltaT, nSpecie_, ddeltaTChem, dYvf, Jss, buffer_span, ode);
        thrust::for_each(thrust::device,
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(nCells),
                         op);
    }
    */



    singleCell op(deltaT, nSpecie_, ddeltaTChem, dYvf, buffer_span, solver_);


    thrust::for_each(thrust::device,
                     thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(nCells),
                     op);

    return std::make_pair(toStdVector(dYvf_arr), toStdVector(ddeltaTChem_arr));
}

std::tuple<std::vector<gScalar>, std::vector<gScalar>, gScalar>
GpuKernelEvaluator::computeRR(gScalar                    deltaT,
                              gScalar                    deltaTChemMax,
                              const std::vector<gScalar> rho,
                              const std::vector<gScalar> deltaTChem,
                              const std::vector<gScalar> Y) const {

    const gLabel nCells = rho.size();

    auto pair          = computeYNew(deltaT, deltaTChemMax, deltaTChem, Y);
    auto YNew_arr      = std::get<0>(pair);
    auto deltaTChemNew = std::get<1>(pair);

    auto YNew = make_mdspan(YNew_arr, extents<2>{nCells, nEqns_});
    auto Y0   = make_mdspan(Y, extents<2>{nCells, nEqns_});

    std::vector<gScalar> RR_arr(nCells * nSpecie_);
    auto                 RR = make_mdspan(RR_arr, extents<2>{nCells, nSpecie_});

    for (gLabel j = 0; j < nCells; ++j) {
        for (gLabel i = 0; i < nSpecie_; ++i) {

            RR(j, i) = rho[j] * (YNew(j, i) - Y0(j, i)) / deltaT;
        }
    }

    gScalar deltaTMin =
        *std::min_element(deltaTChemNew.begin(), deltaTChemNew.end());

    for (auto& e : deltaTChemNew) { e = std::min(e, deltaTChemMax); }

    return std::make_tuple(RR_arr, deltaTChemNew, deltaTMin);
}

} // namespace FoamGpu